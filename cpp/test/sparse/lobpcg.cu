/*
 * Copyright (c) 2019-2022, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>

#include <raft/core/device_mdspan.hpp>
#include <raft/core/handle.hpp>
#include <raft/sparse/linalg/add.cuh>
#include <raft/sparse/solver/lobpcg.cuh>
#include <raft/spectral/matrix_wrappers.hpp>

#include "../test_utils.h"
#include <raft/util/cudart_utils.hpp>

#include <iostream>
#include <limits>

namespace raft {
namespace sparse {

template <typename math_t, typename idx_t>
struct CSRMatrixVal {
  std::vector<idx_t> row_ind;
  std::vector<idx_t> row_ind_ptr;
  std::vector<math_t> values;
};

template <typename math_t, typename idx_t>
struct LOBPCGInputs {
  CSRMatrixVal<math_t, idx_t> matrix_a;
  std::vector<math_t> init_eigvecs;
  std::vector<math_t> exp_eigvals;
  std::vector<math_t> exp_eigvecs;
  idx_t n_components;
};

template <typename math_t, typename idx_t>
class LOBPCGTest : public ::testing::TestWithParam<LOBPCGInputs<math_t, idx_t>> {
 public:
  LOBPCGTest()
    : params(::testing::TestWithParam<LOBPCGInputs<math_t, idx_t>>::GetParam()),
      stream(handle.get_stream()),
      ind_a(params.matrix_a.row_ind.size(), stream),
      ind_ptr_a(params.matrix_a.row_ind_ptr.size(), stream),
      values_a(params.matrix_a.row_ind_ptr.size(), stream),
      exp_eigvals(params.exp_eigvals.size(), stream),
      exp_eigvecs(params.exp_eigvecs.size(), stream),
      act_eigvals(params.exp_eigvals.size(), stream),
      act_eigvecs(params.exp_eigvecs.size(), stream)
  {
  }

 protected:
  void SetUp() override
  {
    n_rows_a = params.matrix_a.row_ind.size() - 1;
    nnz_a    = params.matrix_a.row_ind_ptr.size();
  }

  void test_b_orthonormalize()
  {
    idx_t n_rows_v = n_rows_a;
    idx_t n_features_v = params.n_components;
    raft::update_device(act_eigvecs.data(), params.init_eigvecs.data(), act_eigvecs.size(), stream);
    auto v = raft::make_device_matrix_view<math_t, idx_t, raft::col_major>(
      act_eigvecs.data(), n_rows_v, n_features_v);
    auto bv = raft::make_device_matrix<math_t, idx_t, raft::col_major>(handle, n_rows_v, n_features_v);
    auto vbv = raft::make_device_matrix<math_t, idx_t, raft::col_major>(handle, n_features_v, n_features_v);
    raft::sparse::solver::detail::b_orthonormalize(handle,
      v,
      bv.view(),
      std::nullopt,
      std::make_optional(vbv.view()),
      std::nullopt,
      true
    );
    std::vector<math_t> vbv_inv_expected{0.76298383, 0.0, -1.20276028, 1.0791533};
    std::vector<math_t> vbv_inv_actual(4);
    raft::copy(vbv_inv_actual.data(), vbv.data_handle(), vbv_inv_actual.size(), stream);

    RAFT_CUDA_TRY(hipStreamSynchronize(stream));
    ASSERT_TRUE(hostVecMatch(vbv_inv_expected, vbv_inv_actual, raft::CompareApprox<math_t>(0.0001)));
  }

  void Run()
  {
    test_b_orthonormalize();
    raft::update_device(ind_a.data(), params.matrix_a.row_ind.data(), n_rows_a, stream);
    raft::update_device(ind_ptr_a.data(), params.matrix_a.row_ind_ptr.data(), nnz_a, stream);
    raft::update_device(values_a.data(), params.matrix_a.values.data(), nnz_a, stream);

    raft::update_device(act_eigvecs.data(), params.init_eigvecs.data(), act_eigvecs.size(), stream);

    auto matA = raft::spectral::matrix::sparse_matrix_t(
      handle, ind_ptr_a.data(), ind_a.data(), values_a.data(), n_rows_a, n_rows_a, nnz_a);
    raft::sparse::solver::lobpcg(
      handle,
      matA,
      raft::make_device_matrix_view<math_t, idx_t, raft::col_major>(
        act_eigvecs.data(), n_rows_a, params.n_components),
      raft::make_device_vector_view<math_t, idx_t>(act_eigvals.data(), n_rows_a));

    std::vector<math_t> X_CPU(n_rows_a * params.n_components);
    std::vector<math_t> W_CPU(n_rows_a);
    raft::copy(X_CPU.data(), act_eigvecs.data(), X_CPU.size(), stream);
    raft::copy(W_CPU.data(), act_eigvals.data(), W_CPU.size(), stream);
    ASSERT_TRUE(raft::devArrMatch<math_t>(
      exp_eigvecs.data(), act_eigvecs.data(), exp_eigvecs.size(), raft::CompareApprox<math_t>(0.0001), stream));
    ASSERT_TRUE(raft::devArrMatch<math_t>(
      exp_eigvals.data(), act_eigvals.data(), exp_eigvals.size(), raft::CompareApprox<math_t>(0.0001), stream));
  }

 protected:
  raft::handle_t handle;
  hipStream_t stream;

  LOBPCGInputs<math_t, idx_t> params;
  idx_t n_rows_a, nnz_a;
  rmm::device_uvector<idx_t> ind_a, ind_ptr_a;
  rmm::device_uvector<math_t> values_a, exp_eigvals, exp_eigvecs, act_eigvals, act_eigvecs;
};

using LOBPCGTestF = LOBPCGTest<float, int>;
TEST_P(LOBPCGTestF, Result) { Run(); }

using LOBPCGTestD = LOBPCGTest<double, int>;
TEST_P(LOBPCGTestD, Result) { Run(); }

const std::vector<LOBPCGInputs<float, int>> lobpcg_inputs_f = {
  {{{0, 4, 10, 14, 19, 24, 28},
    {0, 2, 3, 5, 0, 1, 2, 3, 4, 5, 0, 2, 3, 5, 1, 2, 3, 4, 5, 0, 2, 3, 4, 5, 0, 2, 3, 4},
    {0.37911922, 0.11567201, 0.5135106,  0.08968836, 0.73450965, 0.26432646, 0.21985123,
     0.74888277, 0.34753734, 0.11204864, 0.82902676, 0.53023521, 0.24047095, 0.37913592,
     0.60975031, 0.60746519, 0.96833343, 0.30845102, 0.88653955, 0.43530847, 0.32938903,
     0.82477561, 0.20858375, 0.24755519, 0.23677223, 0.73957246, 0.09050876, 0.86530489}},
   {0.08319983,
    0.17758466,
    0.93301819,
    0.67171826,
    0.19967821,
    0.30873092,
    0.35005079,
    0.56035486,
    0.64176631,
    0.93904784,
    0.38935935,
    0.97182089},
   {2.61153278, 0.85782948},
   {-0.38272064,
    -0.25160901,
    -0.48684676,
    -0.50752949,
    -0.43005954,
    -0.33265696,
    -0.39778489,
    0.2539629,
    -0.37506003,
    0.72637041,
    0.02727131,
    -0.32900198},
   2}};
const std::vector<LOBPCGInputs<double, int>> lobpcg_inputs_d = {
  {{{0, 4, 10, 14, 19, 24, 28},
    {0, 2, 3, 5, 0, 1, 2, 3, 4, 5, 0, 2, 3, 5, 1, 2, 3, 4, 5, 0, 2, 3, 4, 5, 0, 2, 3, 4},
    {0.37911922, 0.11567201, 0.5135106,  0.08968836, 0.73450965, 0.26432646, 0.21985123,
     0.74888277, 0.34753734, 0.11204864, 0.82902676, 0.53023521, 0.24047095, 0.37913592,
     0.60975031, 0.60746519, 0.96833343, 0.30845102, 0.88653955, 0.43530847, 0.32938903,
     0.82477561, 0.20858375, 0.24755519, 0.23677223, 0.73957246, 0.09050876, 0.86530489}},
   {0.08319983,
    0.17758466,
    0.93301819,
    0.67171826,
    0.19967821,
    0.30873092,
    0.35005079,
    0.56035486,
    0.64176631,
    0.93904784,
    0.38935935,
    0.97182089},
   {2.61153278, 0.85782948},
   {-0.38272064,
    -0.25160901,
    -0.48684676,
    -0.50752949,
    -0.43005954,
    -0.33265696,
    -0.39778489,
    0.2539629,
    -0.37506003,
    0.72637041,
    0.02727131,
    -0.32900198},
   2}};

INSTANTIATE_TEST_CASE_P(LOBPCGTest, LOBPCGTestF, ::testing::ValuesIn(lobpcg_inputs_f));
INSTANTIATE_TEST_CASE_P(LOBPCGTest, LOBPCGTestD, ::testing::ValuesIn(lobpcg_inputs_d));

}  // namespace sparse
}  // namespace raft

/*

a=cupyx.scipy.sparse.random(6,6, 0.8,'csr')
a.indptr = array([ 0,  4, 10, 14, 19, 24, 28], dtype=int32)

a.indices = array([0, 2, 3, 5, 0, 1, 2, 3, 4, 5, 0, 2, 3, 5, 1, 2, 3, 4, 5, 0, 2, 3,
       4, 5, 0, 2, 3, 4], dtype=int32)

a.data = array([0.37911922, 0.11567201, 0.5135106 , 0.08968836, 0.73450965,
       0.26432646, 0.21985123, 0.74888277, 0.34753734, 0.11204864,
       0.82902676, 0.53023521, 0.24047095, 0.37913592, 0.60975031,
       0.60746519, 0.96833343, 0.30845102, 0.88653955, 0.43530847,
       0.32938903, 0.82477561, 0.20858375, 0.24755519, 0.23677223,
       0.73957246, 0.09050876, 0.86530489])

a.todense() = 
np.matrix([[0.37911922, 0.        , 0.11567201, 0.5135106 , 0.        , 0.08968836],
        [0.73450965, 0.26432646, 0.21985123, 0.74888277, 0.34753734, 0.11204864],
        [0.82902676, 0.        , 0.53023521, 0.24047095, 0.        , 0.37913592],
        [0.        , 0.60975031, 0.60746519, 0.96833343, 0.30845102, 0.88653955],
        [0.43530847, 0.        , 0.32938903, 0.82477561, 0.20858375, 0.24755519],
        [0.23677223, 0.        , 0.73957246, 0.09050876, 0.86530489, 0.        ]])
x = np.random.rand(6,2)
x = np.array([[0.08319983, 0.35005079],
           [0.17758466, 0.56035486],
           [0.93301819, 0.64176631],
           [0.67171826, 0.93904784],
           [0.19967821, 0.38935935],
           [0.30873092, 0.97182089]])

lobpcg(a, x) =  (array([2.61153278, 0.85782948]),
                array([[-0.38272064, -0.39778489],
                        [-0.25160901,  0.2539629 ],
                        [-0.48684676, -0.37506003],
                        [-0.50752949,  0.72637041],
                        [-0.43005954,  0.02727131],
                        [-0.33265696, -0.32900198]]))
 */